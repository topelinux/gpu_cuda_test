#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define DATA_SIZE 1048576
#define THREAD_NUM 256
#define BLOCK_NUM 32

int data[DATA_SIZE];

//初始化CUDA
bool InitCUDA(){
    int count;
    
    hipGetDeviceCount(&count);
    if(count == 0){
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for(int i = 0; i<count;i++){
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess){
            if(prop.major >= 1){
                break;
            }
        }
    }
    
    if(i == count){
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}

//创建0-9的随机数
void GenerateNumbers(int *number, int size){
    for(int i = 0; i < size; i++){
        number[i] = rand() % 10;
    }
}

//显示晶片上执行
__global__ static void sumOfSquares(int *num, int* result, clock_t* time){
    extern __shared__ int shared[];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    int offset = THREAD_NUM / 2;
    int sum = 0;
    int i;
    clock_t start;

    if (tid == 0) time[bid] = clock();

    shared[tid] = 0;
    for(i = tid + bid * THREAD_NUM; i < DATA_SIZE; i += THREAD_NUM * BLOCK_NUM) {
        shared[tid] += num[i]*num[i];
    }
    //__syncthreads();
    //while (offset > 0) { 
    //  if (tid < offset) {
    //    shared[tid] += shared[tid + offset];
    //  }
    //  offset >>= 1;
    //  __syncthreads();
    //}
     if(tid < 128) { shared[tid] += shared[tid + 128]; }
    __syncthreads();
    if(tid < 64) { shared[tid] += shared[tid + 64]; }
    __syncthreads();
    if(tid < 32) { shared[tid] += shared[tid + 32]; }
    __syncthreads();
    if(tid < 16) { shared[tid] += shared[tid + 16]; }
    __syncthreads();
    if(tid < 8) { shared[tid] += shared[tid + 8]; }
    __syncthreads();
    if(tid < 4) { shared[tid] += shared[tid + 4]; }
    __syncthreads();
    if(tid < 2) { shared[tid] += shared[tid + 2]; }
    __syncthreads();
    if(tid < 1) { shared[tid] += shared[tid + 1]; }
    __syncthreads();
    if (tid == 0) {
      result[bid] = shared[0];
      time[bid + BLOCK_NUM] = clock();
    }
}


int main(){
    if(!InitCUDA()){
        return 0;
    }
    
    printf("CUDA initialized.\n");

    GenerateNumbers(data,DATA_SIZE);
    int* gpudata, *result;
    clock_t* time;
    hipMalloc((void**) &gpudata,sizeof(int) * DATA_SIZE);
    hipMalloc((void**) &result,sizeof(int) * THREAD_NUM);
    hipMalloc((void**) &time, sizeof(clock_t) * BLOCK_NUM * 2);
    //从主记忆体复制到显示记忆体，所以使用 cudaMemcpyHostToDevice。
    //如果是从显示记忆体复制到主记忆体，则使用 cudaMemcpyDeviceToHost
    hipMemcpy(gpudata, data,sizeof(int) * DATA_SIZE,
        hipMemcpyHostToDevice);

    //执行函数语法：
    //函数名称<<<block 数目, thread 数目, shared memory 大小>>>(参数);
    sumOfSquares<<<BLOCK_NUM, THREAD_NUM, THREAD_NUM * sizeof(int)>>>(gpudata,result,time);

    int sum[THREAD_NUM * BLOCK_NUM];
    clock_t time_used[BLOCK_NUM * 2];
    hipMemcpy(&sum, result, sizeof(int) * BLOCK_NUM, hipMemcpyDeviceToHost);
    hipMemcpy(&time_used, time, sizeof(clock_t) * BLOCK_NUM * 2, hipMemcpyDeviceToHost);
    hipFree(gpudata);
    hipFree(result);
    hipFree(time);

    clock_t min_start, max_end;
    min_start = time_used[0];
    max_end = time_used[BLOCK_NUM];
    for(int i = 1; i < BLOCK_NUM; i++) {
        if(min_start > time_used[i])
            min_start = time_used[i];
        if(max_end < time_used[i + BLOCK_NUM])
            max_end = time_used[i + BLOCK_NUM];
    }

    int final_sum = 0;
    for(int i = 0; i < BLOCK_NUM; i++) {
        final_sum += sum[i];
    }
    printf("sum: %d  time: %d\n", final_sum, max_end - min_start);

    final_sum = 0;
    for(int i = 0; i < DATA_SIZE; i++) {
        final_sum += data[i] * data[i];
    }
    printf("sum (CPU): %d\n", final_sum);

    return 0;
}
